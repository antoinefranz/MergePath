#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NB 1
#define NTPB 2

#define SIZE 16


void testCUDA(hipError_t error, const char *file, int line)  {
   if (error != hipSuccess) {
      printf("There is an error in file %s at line %d : %d\n", file, line, error);
       exit(EXIT_FAILURE);
   } 
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

#define MAX NB*NTPB
 
typedef struct edge
{
    int u,v,w;
}edge;
 
typedef struct edgelist
{
    edge data[MAX];
    int n;
}edgelist;
 
edgelist elist;
 
//int G[6][6] = {{0,3,1,6,0,0},{3,0,5,0,3,0},{1,5,0,5,6,4},{6,0,5,0,0,2},{0,3,6,0,0,6},{0,0,4,2,6,0}},n;
int G[4][4] = {{0,1,0,0},{1,1,1,3},{0,1,0,2},{0,3,2,0}},n;
//int G[8][8] = {{0,3,1,6,0,0,0,0},
              {3,0,5,0,3,0,3,0},
              {1,5,0,5,6,4,6,0},
              {6,0,5,0,0,2,0,2},
              {0,3,6,0,0,6,0,6},
              {0,0,4,2,6,0,6,0},
              {0,3,6,0,0,6,0,0},
              {0,0,4,2,6,0,0,0}},n;
edgelist spanlist;
 
void kruskal();
int find(int belongs[],int vertexno);
void union1(int belongs[],int c1,int c2);
void print();

__device__ int Ad[NB*NTPB+1];
__device__ int Bd[NB*NTPB+1];

__device__ void printRange(int idx,edge* t, int d,int f){
   
   if(idx == 1){
       printf("\t[");
      for (int i = d; i < f; ++i)
      {
         printf("%d, ",t[i].w);
      }
      printf("]\n");   
   }

   
}


/*------ @param : int 
         idx   : indice du thread qui entre dans la fonction
         C     : tableau de résultat 
         A     : tableau d'entrée
         ab    : indice de debut pour A  
         ad    : indice de fin pour A
         B     : tableau d'entrée
         bb    : indice de début pour B
         bd    : indide de fin pour B


*/
__device__ void merge(int idx,edge* C,edge* A, int ab,int ad,edge* B, int bb, int bd){
   int start = ab+bb;
   int i = 0;
   while(1){
      if(ab >= ad && bb >= bd){

         return;
      }else if(ab == ad){
         C[start+i] = B[bb];
         bb++;
      }else if(bb == bd){
         C[start+i] = A[ab];
         ab++;
      }else if( A[ab].w > B[bb].w){
         C[start+i] = B[bb];
         bb += 1; 
      }else{
         C[start+i] = A[ab];
         ab++;
      }
      i++;
   }
}


/*------ @param : 
            C : fusion de A et B trié
            A : tableau d' entrée trié de lenR éléments
            B : tableau d' entrée trié de lenR éléments
            len* : longeur du talbeau *
            nbT : nombre de thread dans la fonction 

*/
__device__ void mergePath(edge* C,edge* A,int lenA,edge *B,int lenB,int nbT){
   
   int idx = (threadIdx.x + blockIdx.x * blockDim.x)% nbT +1;

   if(idx <= nbT){
 
      Ad[idx-1] = 0;
      Bd[idx-1] = 0;
      if(idx == 1){
         Ad[nbT] = lenA;
         Bd[nbT] = lenB;
       
      
      }
      

      int index = (idx*(lenA+lenB))/(nbT);
      int atop = (index > lenA)? lenA-1: index;
      int btop = (index > lenA)?  index - (lenA-1) : 0;
      int abot = btop;
      int offset;
      int ai;
      int bi;
      int flag = 0;
      while(!flag && idx != nbT){
         offset = (int)floor(((float)(atop - abot))/(float)(2));
         ai = atop - offset;
         bi = btop + offset;
        
         if( bi == 0 || ai >= lenA || A[ai].w > B[bi-1].w){
            if(A[ai-1].w <= B[bi].w)
            {
               Ad[idx] = ai;
               Bd[idx] = bi;
               flag = 1;
            }
            else{
               atop = ai - 1;
               btop = bi + 1;
            }
         }
         else{
            abot = ai + 1;
         }
      }
      __syncthreads();
     // int size = (lenA+lenB)/(nbT);
      for (int i = 1; i <= nbT; ++i)
      {
         merge(idx,C,A,Ad[i-1],Ad[i],B,Bd[i-1],Bd[i]);
      }
   }
}


__device__ void mergeTab(int* C,int* A,int a,int* B,int b)
{
   int start = 0;
   int i = 0;
   int a_ = 0,b_ = 0;
   while(1){
      if(a_ >= a && b_ >= b){
         return;
      }else if(a_ == a){
         C[start+i] = B[b_];
         b_++;
      }else if(b_ == b){
         C[start+i] = A[a_];
         a_++;
      }else if( A[a_] > B[b_]){
         C[start+i] = B[b_];
         b_ += 1; 
      }else{
         C[start+i] = A[a_];
         a_++;
      }
      i++;
   }
}


/*------ @param : 
            R : tableau de sortie trié de lenR éléments
            S : tableau d' entrée trié de lenR éléments
            lenR : longeur des tableaux

*/
__global__ void sort(edge *R,edge* S,int lenR){
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   int size = lenR/(NB*NTPB);
   edge tmp;
   int turn = 1;
   if(lenR < 2*NB*NTPB){
      size = 2;
   }
   printf("%d : size\n",size );
   //création de tableau trié de "size" éléments
   for (int j = 0; j < size; j++)
   {
      for (int k = j; k < size; k++)
      {
         if( R[j+idx*size].w > R[k+idx*size].w){
            tmp = R[j+idx*size];
            R[j+idx*size] = R[k+idx*size];
            R[k+idx*size] = tmp;
         }
      }
   }
   //optimisation ?
   // for(int l = 2 ; l <= size; l *= 2){
   //    for(int m = l ; m <= size ; m *=2){
   //       printf("%d,%d -> mergePath(R+%d,S+%d ,%d, S+%d, %d,%d);\n",idx,m,idx*m,idx*m ,l/2,l/2+idx*m,l/2,1);
   //       mergeTab(R+idx*m,S+idx*m,l/2,S+l/2+idx*m,l/2);   
   //    }
      
   //    for(int i = 0; i < size ; i++){
   //       S[idx*size+i] = R[idx*size+i];

   //    }
   //    // memcpy(S,R,menR);

   // }
  printRange(idx,R,0,lenR);


   for(int i = 0; i < size ; i++){
      S[idx*size+i] = R[idx*size+i];
         
   }
   // memcpy(S+idx*size,R+idx*size,size);
   //__syncthreads();
   turn *=2;
   size *=2;
   idx = idx%((NB*NTPB)/turn);
   while(size <= lenR){
     //printf("%d,%d -> mergePath(R+%d,S+%d ,%d, S+%d, %d,%d);\n",idx,turn,idx*size,idx*size ,size/2,size/2+idx*size,size/2,1);
      mergePath(R+idx*size, S+idx*size ,size/2, S+size/2+idx*size, size/2,turn);
      
      for(int i = 0; i < size ; i++){
         S[idx*size+i] = R[idx*size+i];
         
      }
      // memcpy(S+idx*size,R+idx*size,size);

      turn *=2;
      idx = idx%((NB*NTPB)/turn);
      size *=2 ;

   }
   
}


//affiche un tablrau de "len" élément(s)
void affiche(int* A,int len){
   for (int i = 0; i < len; ++i)
   {
      printf("%d, ",A[i] );
   }
   puts("\b\b  ");
}


int comp (const void * elem1, const void * elem2) 
{
    int f = *((int*)elem1);
    int s_ = *((int*)elem2);
    if (f > s_) return  1;
    if (f < s_) return -1;
    return 0;
}
int np = 4;

int main(int argc,char** argv){

   kruskal();
   print();
   
}

void kruskal()
{
   int belongs[MAX],i,j,cno1,cno2;
   elist.n=0;

   for(i=1;i<np;i++)
     for(j=0;j<i;j++)
     {
         if(G[i][j]!=0)
         {
             elist.data[elist.n].u=i;
             elist.data[elist.n].v=j;
             elist.data[elist.n].w=G[i][j];
             elist.n++;
         }
     }

   //-------------SORT --------------------------
     // Initialisation des tableau
   int *A = (int*)malloc(sizeof(int)*SIZE);
   int* ret = (int*)malloc(sizeof(int)*SIZE);

   // srand(time(NULL));
   for(int i =0 ; i < SIZE ; i++){
      A[i] = rand()%10000+1;
      ret[i] = 0;

   }
   //---------------------------------

   //création des variable sur le GPU
   edge *aGPU,*bGPU;
   testCUDA(hipMalloc(&aGPU,SIZE*sizeof(edge)));
   testCUDA(hipMemcpy(aGPU,elist.data,MAX*sizeof(edge),hipMemcpyHostToDevice));
   testCUDA(hipMalloc(&bGPU,sizeof(elist.data)));
   testCUDA(hipMemcpy(bGPU,elist.data,MAX*sizeof(edge),hipMemcpyHostToDevice));
   //---------------------------------
  // affiche(A,SIZE); // affichage du tableau non trié
   printf("len tab : %d\n",SIZE );

   // gestion du temps 
   int count;
   hipDeviceProp_t prop;
   testCUDA(hipGetDeviceCount(&count));
   testCUDA(hipGetDeviceProperties(&prop, count-1));
   
   float TimerAddOne;                        // GPU timer instructions
   hipEvent_t start, stop;                  // GPU timer instructions
   testCUDA(hipEventCreate(&start));           // GPU timer instructions
   testCUDA(hipEventCreate(&stop));            // GPU timer instructions
   testCUDA(hipEventRecord(start,0));          // GPU timer instructions
   //-----------------


   // lancement de  l'algorithme de tri
   sort<<<NB,NTPB>>>(aGPU,bGPU,SIZE);


   // récupération du temps mis par le GPU
   testCUDA(hipEventRecord(stop,0));           // GPU timer instructions
   testCUDA(hipEventSynchronize(stop));        // GPU timer instructions
   testCUDA(hipEventElapsedTime(&TimerAddOne,     // GPU timer instructions
          start, stop));                     // GPU timer instructions


   // récupération du talbeau sur le CPU
   testCUDA(hipMemcpy(spanlist.data,aGPU,MAX*sizeof(edge),hipMemcpyDeviceToHost));
   printf("\n\n");
   //print();
  // affiche(ret,SIZE);
   printf("GPU Timer for the addition on the GPU of vectors: %f ms\n", 
         TimerAddOne);

   // libération de la memoire
   // free(A);free(ret);
   // testCUDA(hipFree(aGPU));
   // testCUDA(hipFree(bGPU));


   for(i=0;i<np;i++)
     belongs[i]=i;

   spanlist.n=0;

   for(i=0;i<elist.n;i++)
   {
     cno1=find(belongs,elist.data[i].u);
     cno2=find(belongs,elist.data[i].v);
     
     if(cno1!=cno2)
     {
         spanlist.data[spanlist.n]=elist.data[i];
         spanlist.n=spanlist.n+1;
         union1(belongs,cno1,cno2);
     }
   }
}
 
int find(int belongs[],int vertexno)
{
    return(belongs[vertexno]);
}
 
void union1(int belongs[],int c1,int c2)
{
    int i;
    
    for(i=0;i<np;i++)
        if(belongs[i]==c2)
            belongs[i]=c1;
}

void print()
{
    int i,cost=0;
    
    for(i=0;i<spanlist.n;i++)
    {
        printf("\n%d\t%d\t%d",spanlist.data[i].u,spanlist.data[i].v,spanlist.data[i].w);
        cost=cost+spanlist.data[i].w;
    }
 
    printf("\n\nCost of the spanning tree=%d\n",cost);
}

