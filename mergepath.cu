
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NB 2
#define NTPB 4 //Minimum number of threads in this program has to be 64
#define sizeTab
void testCUDA(hipError_t error, const char *file, int line)  {
   if (error != hipSuccess) {
      printf("There is an error in file %s at line %d : %d\n", file, line, error);
       exit(EXIT_FAILURE);
   } 
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

__device__ int Bd[NB*NTPB+1];
__device__ int Ad[NB*NTPB+1];


__device__ void printRange(int idx,int* t, int d,int f){
   
   if(idx == 1){
      for (int i = d; i < f; ++i)
      {
         printf("\ttab[%d] = %d\n", i,t[i]);
      }
      printf("\n");   
   }

      // printf("tab[%d:%d]\", d,f);
   
}


__device__ void merge(int idx,int* C,int* A, int ab,int ad,int* B, int bb, int bd, int pos,int size){
   int start = ab+bb;
   int i = 0;
   while(1){
      if(ab == ad && bb == bd){

         printRange(idx,C,0,16);
         return;
      }else if(ab == ad){
         C[start+i] = B[bb];
         bb++;
      }else if(bb == bd){
         C[start+i] = A[ab];
         ab++;
      }else if( A[ab] > B[bb]){
         C[start+i] = B[bb];
         bb += 1;
      }else{
         C[start+i] = A[ab];
         ab++;
      }
      i++;
   }
}



__global__ void mergePath(int* C,int* A,int lenA,int *B,int lenB){
   int idx = threadIdx.x + blockIdx.x * blockDim.x+1;
   Ad[idx] = 0;
   Bd[idx] = 0;
   if(idx == 0){
      Ad[NB*NTPB] = lenA;
      Bd[NB*NTPB] = lenB;
   }
   
   int index = (idx*(lenA+lenB))/(NB*NTPB);
   int atop = (index > lenA)? lenA-1: index;
   int btop = (index > lenA)?  index - (lenA-1) : 0;
   int abot = btop;
   //int i = 0;
   int offset;
   int ai;
   int bi;
   int flag = 0;
   while(!flag){
      //printf("%d\n",i++ );
      offset = (int)floor(((float)(atop - abot))/(float)(2));
      ai = atop - offset;
      bi = btop + offset;
      // if(idx == 7){
      //    printf("%d : (%d,%d) off: %d, atop %d btop %d abot %d\n",idx,ai,bi,offset,atop,btop,abot );
      // }
      if( bi == 0 || ai >= lenA || A[ai] > B[bi-1]){
         if(A[ai-1] <= B[bi])
         {
            printf("%d : (%d,%d)\n",idx,ai,bi );
            Ad[idx] = ai;
            Bd[idx] = bi;
            flag = 1;
         }
         else{
            atop = ai - 1;
            btop = bi + 1;
         }
      }
      else{
         abot = ai + 1;
      }
   }
   __syncthreads();
   if(idx == 1)
   {
      printRange(1,Ad,0,9);
      printRange(1,Bd,0,9);
   }
   int size = (lenA+lenB)/(NB*NTPB);
   for (int i = 1; i <= NB*NTPB; ++i)
   {
      merge(idx,C,A,Ad[i-1],Ad[i],B,Bd[i-1],Bd[i],i-1,size);
     // printRange(0,C,0,16);
   }
}




void affiche(int* A,int len){
   for (int i = 0; i < len; ++i)
   {
      printf("%d, ",A[i] );
   }
   puts("\b\b  ");
}
#define s 8
int main(int argc,char** argv){
   /*int A[s] = {5,17,11,19,18,15,16,114,119,120,14,112,117,159
                  ,71,66,58,157,42,195,185,15,176,1124,1419
                  ,120,147,112,1147,19,71,646
               };*/
   // int C[2*s] = {0};
   int* C = (int*)calloc(4*s,sizeof(int));
   int A[s] = {1,2,3,4,6,10,11,13};
   int B[s] = {5,7,8,9,12,14,15,16};
   int *aGPU,*bGPU,*cGPU;
   hipMalloc(&aGPU,s*sizeof(int));
   hipMemcpy(aGPU,A,s*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc(&bGPU,s*sizeof(int));
   hipMemcpy(bGPU,B,s*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc(&cGPU,4*s*sizeof(int));
   hipMemset(cGPU,0,2*s*sizeof(int));
 
   mergePath<<<NB,NTPB>>>(cGPU,aGPU,s,bGPU,s);

   testCUDA(hipMemcpy(C,cGPU,2*s*sizeof(int),hipMemcpyDeviceToHost));
   affiche(C,2*s);
}


// __global__ void merge(int* A,int lenTab){
//    int idx = threadIdx.x + blockIdx.x * blockDim.x;
//    int bidx = threadIdx.x;
//    int turn = 1;
//    int end = 0;
//    int size = lenTab/(NB*NTPB);
//    if(size < 2){
//       size = 2;
//    }
//    __shared__ int *sT;
//    //premier tri de size element

//    //tri A[idx*size : (idx+1)*size-1]
//    for (int i = 0; i < size; i++)
//    {
//       sT[bidx+i] = A[idx+i];

//    }
//    for (int j = 0; j < size; j++)
//    {
//       for (int k = j; k < size; k++)
//       {
//          if( sT[j] > sT[k]){
//             int tmp = sT[j];
//             sT[j] = sT[k];
//             sT[k] = sT[j];
//          }
//       }
//    }
//    turn++;
//    size *=2;
//    for (int i = 0; i < size; i++)
//    {
//       // sT[bidx+i] = A[idx+i];
//       A[idx] = sT[bidx];
//    }
//    // while(!end){

//    // }


//    //suite 


// }


// void mergepath(int* C,int *A,int lenA,int* B, int lenB,int nbT){
//    int Adiag[nbT+1] = {0}
//    int Bdiag[nbT+1] = {0}

//    Adiag[nbT] = lenA
//    Bdiag[nbT] = lenB
//    for( int i = 0; )
// }