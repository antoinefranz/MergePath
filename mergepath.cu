
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NB 1
#define NTPB 1024

#define SIZE 2048*512


void testCUDA(hipError_t error, const char *file, int line)  {
   if (error != hipSuccess) {
      printf("There is an error in file %s at line %d : %d\n", file, line, error);
       exit(EXIT_FAILURE);
   } 
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


__device__ int Ad[NB*NTPB+1];
__device__ int Bd[NB*NTPB+1];

__device__ void printRange(int idx,int* t, int d,int f){
   
   if(idx == 1){
       printf("\t[");
      for (int i = d; i < f; ++i)
      {
         printf("%d, ",t[i]);
      }
      printf("]\n");   
   }

   
}


/*------ @param : int 
         idx   : indice du thread qui entre dans la fonction
         C     : tableau de résultat 
         A     : tableau d'entrée
         ab    : indice de debut pour A  
         ad    : indice de fin pour A
         B     : tableau d'entrée
         bb    : indice de début pour B
         bd    : indide de fin pour B


*/
__device__ void merge(int idx,int* C,int* A, int ab,int ad,int* B, int bb, int bd){
   int start = ab+bb;
   int i = 0;
   while(1){
      if(ab >= ad && bb >= bd){

         return;
      }else if(ab == ad){
         C[start+i] = B[bb];
         bb++;
      }else if(bb == bd){
         C[start+i] = A[ab];
         ab++;
      }else if( A[ab] > B[bb]){
         C[start+i] = B[bb];
         bb += 1; 
      }else{
         C[start+i] = A[ab];
         ab++;
      }
      i++;
   }
}


/*------ @param : 
            C : fusion de A et B trié
            A : tableau d' entrée trié de lenR éléments
            B : tableau d' entrée trié de lenR éléments
            len* : longeur du talbeau *
            nbT : nombre de thread dans la fonction 

*/
__device__ void mergePath(int* C,int* A,int lenA,int *B,int lenB,int nbT){
   
   int idx = (threadIdx.x + blockIdx.x * blockDim.x)% nbT +1;

   if(idx <= nbT){
 
      Ad[idx-1] = 0;
      Bd[idx-1] = 0;
      if(idx == 1){
         Ad[nbT] = lenA;
         Bd[nbT] = lenB;
       
      
      }
      

      int index = (idx*(lenA+lenB))/(nbT);
      int atop = (index > lenA)? lenA-1: index;
      int btop = (index > lenA)?  index - (lenA-1) : 0;
      int abot = btop;
      int offset;
      int ai;
      int bi;
      int flag = 0;
      while(!flag && idx != nbT){
         offset = (int)floor(((float)(atop - abot))/(float)(2));
         ai = atop - offset;
         bi = btop + offset;
        
         if( bi == 0 || ai >= lenA || A[ai] > B[bi-1]){
            if(A[ai-1] <= B[bi])
            {
               Ad[idx] = ai;
               Bd[idx] = bi;
               flag = 1;
            }
            else{
               atop = ai - 1;
               btop = bi + 1;
            }
         }
         else{
            abot = ai + 1;
         }
      }
      __syncthreads();
     // int size = (lenA+lenB)/(nbT);
      for (int i = 1; i <= nbT; ++i)
      {
         merge(idx,C,A,Ad[i-1],Ad[i],B,Bd[i-1],Bd[i]);
      }
   }
}


__device__ void mergeTab(int* C,int* A,int a,int* B,int b)
{
   int start = 0;
   int i = 0;
   int a_ = 0,b_ = 0;
   while(1){
      if(a_ >= a && b_ >= b){
         return;
      }else if(a_ == a){
         C[start+i] = B[b_];
         b_++;
      }else if(b_ == b){
         C[start+i] = A[a_];
         a_++;
      }else if( A[a_] > B[b_]){
         C[start+i] = B[b_];
         b_ += 1; 
      }else{
         C[start+i] = A[a_];
         a_++;
      }
      i++;
   }
}


/*------ @param : 
            R : tableau de sortie trié de lenR éléments
            S : tableau d' entrée trié de lenR éléments
            lenR : longeur des tableaux

*/
__global__ void sort(int *R,int* S,int lenR){
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   int size = lenR/(NB*NTPB);
   int tmp;
   int turn = 1;
   if(lenR < 2*NB*NTPB){
      size = 2;
   }
   printf("%d : size\n",size );
   //création de tableau trié de "size" éléments
   for (int j = 0; j < size; j++)
   {
      for (int k = j; k < size; k++)
      {
         if( R[j+idx*size] > R[k+idx*size]){
            tmp = R[j+idx*size];
            R[j+idx*size] = R[k+idx*size];
            R[k+idx*size] = tmp;
         }
      }
   }
   //optimisation ?
   // for(int l = 2 ; l <= size; l *= 2){
   //    for(int m = l ; m <= size ; m *=2){
   //       printf("%d,%d -> mergePath(R+%d,S+%d ,%d, S+%d, %d,%d);\n",idx,m,idx*m,idx*m ,l/2,l/2+idx*m,l/2,1);
   //       mergeTab(R+idx*m,S+idx*m,l/2,S+l/2+idx*m,l/2);   
   //    }
      
   //    for(int i = 0; i < size ; i++){
   //       S[idx*size+i] = R[idx*size+i];

   //    }
   //    // memcpy(S,R,menR);

   // }
  printRange(idx,R,0,lenR);


   for(int i = 0; i < size ; i++){
      S[idx*size+i] = R[idx*size+i];
         
   }
   // memcpy(S+idx*size,R+idx*size,size);
   //__syncthreads();
   turn *=2;
   size *=2;
   idx = idx%((NB*NTPB)/turn);
   while(size <= lenR){
     //printf("%d,%d -> mergePath(R+%d,S+%d ,%d, S+%d, %d,%d);\n",idx,turn,idx*size,idx*size ,size/2,size/2+idx*size,size/2,1);
      mergePath(R+idx*size, S+idx*size ,size/2, S+size/2+idx*size, size/2,turn);
      
      for(int i = 0; i < size ; i++){
         S[idx*size+i] = R[idx*size+i];
         
      }
      // memcpy(S+idx*size,R+idx*size,size);

      turn *=2;
      idx = idx%((NB*NTPB)/turn);
      size *=2 ;

   }
   
}


//affiche un tablrau de "len" élément(s)
void affiche(int* A,int len){
   for (int i = 0; i < len; ++i)
   {
      printf("%d, ",A[i] );
   }
   puts("\b\b  ");
}


int comp (const void * elem1, const void * elem2) 
{
    int f = *((int*)elem1);
    int s_ = *((int*)elem2);
    if (f > s_) return  1;
    if (f < s_) return -1;
    return 0;
}

int main(int argc,char** argv){

   // Initialisation des tableau
   int *A = (int*)malloc(sizeof(int)*SIZE);
   int* ret = (int*)malloc(sizeof(int)*SIZE);

   // srand(time(NULL));
   for(int i =0 ; i < SIZE ; i++){
      A[i] = rand()%10000+1;
      ret[i] = 0;

   }
   //---------------------------------

   //création des variable sur le GPU
   int *aGPU,*bGPU;
   testCUDA(hipMalloc(&aGPU,SIZE*sizeof(int)));
   testCUDA(hipMemcpy(aGPU,A,SIZE*sizeof(int),hipMemcpyHostToDevice));
   testCUDA(hipMalloc(&bGPU,SIZE*sizeof(int)));
   testCUDA(hipMemcpy(bGPU,A,SIZE*sizeof(int),hipMemcpyHostToDevice));
   //---------------------------------
   affiche(A,SIZE); // affichage du tableau non trié
   printf("len tab : %d\n",SIZE );

   // gestion du temps 
   int count;
   hipDeviceProp_t prop;
   testCUDA(hipGetDeviceCount(&count));
   testCUDA(hipGetDeviceProperties(&prop, count-1));
   
   float TimerAddOne;                        // GPU timer instructions
   hipEvent_t start, stop;                  // GPU timer instructions
   testCUDA(hipEventCreate(&start));           // GPU timer instructions
   testCUDA(hipEventCreate(&stop));            // GPU timer instructions
   testCUDA(hipEventRecord(start,0));          // GPU timer instructions
   //-----------------


   // lancement de  l'algorithme de tri
   sort<<<NB,NTPB>>>(aGPU,bGPU,SIZE);


   // récupération du temps mis par le GPU
   testCUDA(hipEventRecord(stop,0));           // GPU timer instructions
   testCUDA(hipEventSynchronize(stop));        // GPU timer instructions
   testCUDA(hipEventElapsedTime(&TimerAddOne,     // GPU timer instructions
          start, stop));                     // GPU timer instructions


   // récupération du talbeau sur le CPU
   testCUDA(hipMemcpy(ret,aGPU,SIZE*sizeof(int),hipMemcpyDeviceToHost));
   printf("\n\n");
   affiche(ret,SIZE);
   printf("GPU Timer for the addition on the GPU of vectors: %f ms\n", 
         TimerAddOne);

   // libération de la memoire
   free(A);free(ret);
   testCUDA(hipFree(aGPU));
   testCUDA(hipFree(bGPU));
}

