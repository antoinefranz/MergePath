
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NB 2
#define NTPB 4 
#define sizeTab
void testCUDA(hipError_t error, const char *file, int line)  {
   if (error != hipSuccess) {
      printf("There is an error in file %s at line %d : %d\n", file, line, error);
       exit(EXIT_FAILURE);
   } 
}

#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))


__device__ int Ad[NB*NTPB+1];
__device__ int Bd[NB*NTPB+1];

__device__ void printRange(int idx,int* t, int d,int f){
   
   if(idx == 1){
       printf("\t[");
      for (int i = d; i < f; ++i)
      {
         printf("%d, ",t[i]);
      }
      printf("]\n");   
   }

      // printf("tab[%d:%d]\", d,f);
   
}


__device__ void merge(int idx,int* C,int* A, int ab,int ad,int* B, int bb, int bd, int pos,int size){
   int start = ab+bb;
   int i = 0;
  // printf("->%d\n", ad-ab + bd-bb);
   while(1){
      if(ab >= ad && bb >= bd){

        // printRange(idx,C,0,16);
         return;
      }else if(ab == ad){
         C[start+i] = B[bb];
         bb++;
      }else if(bb == bd){
         C[start+i] = A[ab];
         ab++;
      }else if( A[ab] > B[bb]){
         C[start+i] = B[bb];
         bb += 1; 
      }else{
         C[start+i] = A[ab];
         ab++;
      }
      i++;
   }
}



__device__ void mergePath(int* C,int* A,int lenA,int *B,int lenB,int nbT){
   
   int idx = (threadIdx.x + blockIdx.x * blockDim.x)% nbT +1;

   if(idx <= nbT){

      //C[idx-1] = idx;
      // Ad[0] = 0;
      // Bd[0] = 0;
      int test = 0;
      printf("\t\t%d  = idx (%d , %d et %d , %d\n",idx,A[0] ,A[1] , B[0], B[1]   );
      if(A[0] == 2 && A[1] == 9 && lenA == 2 && B[0] == 7 && B[1] == 10 ){
         test =1;
         printf("!!!!!!!!!!! %d\n",idx);
      }
      Ad[idx-1] = 0;
      Bd[idx-1] = 0;
      if(idx == 1){
      //    // printf("\n");
         Ad[nbT] = lenA;
         Bd[nbT] = lenB;
       
      
      }
      

      int index = (idx*(lenA+lenB))/(nbT);
      int atop = (index > lenA)? lenA-1: index;
      int btop = (index > lenA)?  index - (lenA-1) : 0;
      int abot = btop;
      //int i = 0;
      int offset;
      int ai;
      int bi;
      int flag = 0;
      while(!flag && idx != nbT){
         //printf("%d\n",i++ );
         offset = (int)floor(((float)(atop - abot))/(float)(2));
         ai = atop - offset;
         bi = btop + offset;
         if(test){
            printf("%d : (%d,%d) off: %d, atop %d btop %d abot %d\n %d > %d et %d <= %d ?  \n",idx,ai,bi,offset,atop,btop,abot,A[ai], B[bi-1] ,A[ai-1],B[bi]);
         }
         if( bi == 0 || ai >= lenA || A[ai] > B[bi-1]){
            if(A[ai-1] <= B[bi])
            {
              printf("%d : (%d,%d)\n",idx,ai,bi );
               Ad[idx] = ai;
               Bd[idx] = bi;
               flag = 1;
            }
            else{
               atop = ai - 1;
               btop = bi + 1;
            }
         }
         else{
            abot = ai + 1;
         }
      }
      __syncthreads();
         if(/*idx == 1 && */test){
            printf("plop\n");
         printRange(1,Ad,0,9);
         printRange(1,Bd,0,9);
      }
      int size = (lenA+lenB)/(nbT);
      for (int i = 1; i <= nbT; ++i)
      {
         merge(idx,C,A,Ad[i-1],Ad[i],B,Bd[i-1],Bd[i],i-1,size);
        // printRange(0,C,0,16);
      }
   }
}

__global__ void sort(int *R,int* S,int* T,int lenR){
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   int size = lenR/(NB*NTPB);
   int tmp;
   int turn = 1;
 //  printf("debut\n");
   if(2*lenR < NB*NTPB){
   //   printf("inf\n");
   }else{
     // printf("sup %d\n",size);

      for (int j = 0; j < size; j++)
      {
         for (int k = j; k < size; k++)
         {
            if( R[j+idx*size] > R[k+idx*size]){
               tmp = R[j+idx*size];
               R[j+idx*size] = R[k+idx*size];
               R[k+idx*size] = tmp;
            }
         }
      }
      for(int i = 0; i < size ; i++){
         S[idx*size+i] = R[idx*size+i];
            
      }
      printRange(idx,R,0,16);
      //fin premier etape
      __syncthreads();
      turn *=2;
      size *=2;
      //printf("fin premiere etape\n");
      while(size <= lenR){
         if( idx < lenR/(turn*2)){

            printf("%d -> mergePath(R+%d,S+%d ,%d, S+%d, %d,%d);\n",idx,idx*size,idx*size ,size/2,size/2+idx*size,size/2,1);
            printRange(idx,S,4,8);
            mergePath(R+idx*size,/*R+idx*size/2*/S+idx*size ,size/2, /*R+size/2+idx*size/2*/S+size/2+idx*size, size/2,1);
            __syncthreads();
            printRange(idx,R,4,8);
         }
         __syncthreads();
         // mergePath(R+idx*size,/*R+idx*size/2*/S+idx*size/2 ,size/2, /*R+size/2+idx*size/2*/S+size/2+idx*size/2, size/2,turn);
         for(int i = 0; i < size ; i++){
            S[idx*size+i] = R[idx*size+i];
            // T[idx*size+i] = R[idx*size+i];
            
         }
        // printf("traitement de : %d\n", size);
         turn *=2;
         size *=2 ;
         printRange(idx,R,0,16);
         __syncthreads();
      }
   }
}



void affiche(int* A,int len){
   for (int i = 0; i < len; ++i)
   {
      printf("%d, ",A[i] );
   }
   puts("\b\b  ");
}
#define s 512

int comp (const void * elem1, const void * elem2) 
{
    int f = *((int*)elem1);
    int s_ = *((int*)elem2);
    if (f > s_) return  1;
    if (f < s_) return -1;
    return 0;
}

int main(int argc,char** argv){
   // /*int A[s] = {5,17,11,19,18,15,16,114,119,120,14,112,117,159
   //                ,71,66,58,157,42,195,185,15,176,1124,1419
   //                ,120,147,112,1147,19,71,646
   //             };*/
   // // int C[2*s] = {0};
   // srand(time(NULL));
   // int* C = (int*)calloc(4*s,sizeof(int));
   // // int A[s] = {1,2,3,4,6,10,11,13};
   // // int B[s] = {5,7,8,9,12,14,15,16};
   // int *A = (int*)malloc(sizeof(int)*s);
   // // int *B = (int*)malloc(sizeof(int)*s);

   // int* ret = (int*)malloc(sizeof(int)*s);
   // for(int i =0 ; i < s ; i++){
   //    A[i] = rand()%1000+1;
   //    // B[i] = rand()%1000+1;
   //    ret[i] = 0;

   // }

   // int *aGPU,*bGPU,*cGPU;
   // cudaMalloc(&aGPU,s*sizeof(int));
   // cudaMemcpy(aGPU,A,s*sizeof(int),cudaMemcpyHostToDevice);
   
   // cudaMalloc(&bGPU,s*sizeof(int));
   // cudaMemcpy(bGPU,A,s*sizeof(int),cudaMemcpyHostToDevice);
   
   // cudaMalloc(&cGPU,2*s*sizeof(int));
   // cudaMemcpy(cGPU,A,s*sizeof(int),cudaMemcpyHostToDevice);
   // printf("debut :\n");
   // sort<<<NB,NTPB>>>(aGPU,bGPU,cGPU,s);

   // testCUDA(cudaMemcpy(ret,aGPU,s*sizeof(int),cudaMemcpyDeviceToHost));
   // affiche(ret,s);



   // qsort(A,s,sizeof(int),comp);

   // qsort(B,s,sizeof(int),comp);
   int t[16] = {7,5,1,4,9,2,7,10,11,20,3,8,12,30,17,18};
   affiche(t,16);
   int *tGPU,*uGPU,*vGPU;
   int *rettest = (int*)malloc(sizeof(int)*16);
   hipMalloc(&tGPU,16*sizeof(int));
   hipMemcpy(tGPU,t,16*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc(&uGPU,16*sizeof(int));
   hipMemcpy(uGPU,t,16*sizeof(int),hipMemcpyHostToDevice);
   hipMalloc(&vGPU,16*sizeof(int));
   hipMemcpy(vGPU,t,16*sizeof(int),hipMemcpyHostToDevice);
   

   sort<<<NB,NTPB>>>(tGPU,uGPU,vGPU,16);


   // for(int i = 0 ; i < 4; i++){
   //    mergePath<<<NB,NTPB>>>(tGPU+i*4,uGPU+i*4,2,vGPU+2+i*4,2,2); 
   //    cudaMemcpy(uGPU,tGPU,16*sizeof(int),cudaMemcpyDeviceToDevice);
   //    cudaMemcpy(vGPU,tGPU,16*sizeof(int),cudaMemcpyDeviceToDevice); 
   //    testCUDA(cudaMemcpy(rettest,tGPU,16*sizeof(int),cudaMemcpyDeviceToHost));
   //    affiche(rettest,16);
   // }
   //   for(int i = 0 ; i < 2; i++){
   //    mergePath<<<NB,NTPB>>>(tGPU+i*8,uGPU+i*8,4,vGPU+4+i*8,4,4); 
   //    cudaMemcpy(uGPU,tGPU,16*sizeof(int),cudaMemcpyDeviceToDevice);
   //    cudaMemcpy(vGPU,tGPU,16*sizeof(int),cudaMemcpyDeviceToDevice); 
   //    testCUDA(cudaMemcpy(rettest,tGPU,16*sizeof(int),cudaMemcpyDeviceToHost));
   //    affiche(rettest,16);
   // }
   
   
   testCUDA(hipMemcpy(rettest,tGPU,16*sizeof(int),hipMemcpyDeviceToHost));
   affiche(rettest,16);
}


// __global__ void merge(int* A,int lenTab){
//    int idx = threadIdx.x + blockIdx.x * blockDim.x;
//    int bidx = threadIdx.x;
//    int turn = 1;
//    int end = 0;
//    int size = lenTab/(NB*NTPB);
//    if(size < 2){
//       size = 2;
//    }
//    __shared__ int *sT;
//    //premier tri de size element

//    //tri A[idx*size : (idx+1)*size-1]
//    for (int i = 0; i < size; i++)
//    {
//       sT[bidx+i] = A[idx+i];

//    }
//    for (int j = 0; j < size; j++)
//    {
//       for (int k = j; k < size; k++)
//       {
//          if( sT[j] > sT[k]){
//             int tmp = sT[j];
//             sT[j] = sT[k];
//             sT[k] = sT[j];
//          }
//       }
//    }
//    turn++;
//    size *=2;
//    for (int i = 0; i < size; i++)
//    {
//       // sT[bidx+i] = A[idx+i];
//       A[idx] = sT[bidx];
//    }
//    // while(!end){

//    // }


//    //suite 


// }


// void mergepath(int* C,int *A,int lenA,int* B, int lenB,int nbT){
//    int Adiag[nbT+1] = {0}
//    int Bdiag[nbT+1] = {0}

//    Adiag[nbT] = lenA
//    Bdiag[nbT] = lenB
//    for( int i = 0; )
// }